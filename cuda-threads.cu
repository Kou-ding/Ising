
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void isingCuda(int* currentCuda, int* nextCuda, int n) {
    
    // Calculate the global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        // Calculate the sum of neighboring spins
        int sum = currentCuda[((i - 1 + n) % n)*n + j] +
                currentCuda[((i + 1) % n)*n + j] +
                currentCuda[i*n + (j - 1 + n) % n] +
                currentCuda[i*n + (j + 1) % n] + 
                currentCuda[i*n + j];
        // Update the next state based on the sum
        nextCuda[i*n + j] = sum > 0 ? 1 : -1;
    }
}

void isingSimulation(int n, int k) {
    // Allocate input vectors h_A and h_B in host memory
    int* current = (int*)malloc(n * n * sizeof(int));
    if (current == NULL) {
        printf("current: Memory not available.\n");
        exit(1);
    }

    int* next = (int*)malloc(n * n * sizeof(int));
    if (next == NULL) {
        printf("next: Memory not available.\n");
        exit(1);
    }

    // Allocate vectors in device memory
    int* currentCuda;
    hipMalloc(&currentCuda, n * n * sizeof(int));
    if (currentCuda == NULL) {
        printf("currentCuda: Memory not available.\n");
        exit(1);
    }
    int* nextCuda;
    hipMalloc(&nextCuda, n * n * sizeof(int));
    if (nextCuda == NULL) {
        printf("nextCuda: Memory not available.\n");
        exit(1);
    }

    // Initialize the current state with random values
    srand(1);
    //srand(time(NULL));
    for (int i = 0; i < n * n; i++) {
        current[i] = rand() % 2 == 0 ? -1 : 1;
    }

    // Copy vectors from host memory to device memory
    hipMemcpy(currentCuda, current, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(nextCuda, next, n * n * sizeof(int), hipMemcpyHostToDevice);

   
    // Declare cuda parameters
    int numBlocks = 1; // Number of blocks
    dim3 threadsPerBlock(n, n); // Number of threads per block

    // Perform k iterations
    for (int iter = 0; iter < k; iter++) {
        // Call the kernel function with n blocks and n threads per block
        isingCuda<<<numBlocks, threadsPerBlock>>>(currentCuda, nextCuda, n);

        // Swap the pointers of current and next arrays
        int* temp = currentCuda;
        currentCuda = nextCuda;
        nextCuda = temp;
        
        // Wait for all threads to finish
        hipDeviceSynchronize();
    }
    hipMemcpy(current, currentCuda, n * n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(next, nextCuda, n * n * sizeof(int), hipMemcpyDeviceToHost);
    /* Print the matrix
    printf("|");
    for (int i = 0; i < n * n; i++) {
        if (current[i] == 1){
            printf(" %d|", current[i]);
        } 
        else{
            printf("%d|", current[i]);
        }
        if (((i+1)%n)==0 && i!=0 && i!=n*n-1) printf("\n|");
        if (((i+1)%(n*n))==0 && i!=0) printf("\n");
    }*/
    // Free the memory
    free(current);
    free(next);
    hipFree(currentCuda);
    hipFree(nextCuda);
}

int main() {
    int n; // Size of the Ising model
    int k; // Number of iterations
    printf("Enter the size of the Ising model: ");
    scanf("%d", &n);
    printf("Enter the number of iterations: ");
    scanf("%d", &k);
    
    clock_t start, end;
    double cpu_time_used;

    start = clock();

    isingSimulation(n, k);
    
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time taken: %f seconds\n", cpu_time_used);
    return 0;
}
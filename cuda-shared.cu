
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void isingCuda(int* currentCuda, int* nextCuda, int n, int elementsPerThread) {
    // Declare shared memory
    extern __shared__ int sharedCurrent[];

    // Calculate the global index of the thread
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread loads multiple elements into shared memory
    for (int offset = 0; offset < elementsPerThread; offset++) {
        int idx = globalIdx * elementsPerThread + offset;
        if (idx < n * n) {
            sharedCurrent[idx] = currentCuda[idx];
        }
    }

    // Synchronize to make sure the data is loaded before computation
    __syncthreads();

    // Each thread processes multiple elements
    for (int offset = 0; offset < elementsPerThread; offset++) {
        int idx = globalIdx * elementsPerThread + offset;
        if (idx < n * n) {
            int i = idx / n;
            int j = idx % n;

            // Calculate the sum of neighboring spins using shared memory
            int sum = sharedCurrent[((i - 1 + n) % n)*n + j] +
                    sharedCurrent[((i + 1) % n)*n + j] +
                    sharedCurrent[i*n + (j - 1 + n) % n] +
                    sharedCurrent[i*n + (j + 1) % n] + 
                    sharedCurrent[i*n + j];
            // Update the next state based on the sum
            nextCuda[i*n + j] = sum > 0 ? 1 : -1;
        }
    }
}

void isingSimulation(int n, int k, int numThreads) {
    // Allocate input vectors h_A and h_B in host memory
    int* current = (int*)malloc(n * n * sizeof(int));
    int* next = (int*)malloc(n * n * sizeof(int));

    // Allocate vectors in device memory
    int* currentCuda;
    hipMalloc(&currentCuda, n * n * sizeof(int));
    int* nextCuda;
    hipMalloc(&nextCuda, n * n * sizeof(int));

     // Initialize the current state with random values
     srand(1);
    //srand(time(NULL));
    for (int i = 0; i < n * n; i++) {
        current[i] = rand() % 2 == 0 ? -1 : 1;
    }

    // Copy vectors from host memory to device memory
    hipMemcpy(currentCuda, current, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(nextCuda, next, n * n * sizeof(int), hipMemcpyHostToDevice);

   
    // Declare cuda parameters
    int numBlocks = 1; // Number of blocks
    //dim3 threadsPerBlock(n, n); // Number of threads per block
    // Calculate the number of elements processed by each thread
    int elementsPerThread = (n * n + numThreads - 1) / numThreads;

    // Perform k iterations
    for (int iter = 0; iter < k; iter++) {
        // Call the kernel function with n blocks and n threads per block
        isingCuda<<<numBlocks, numThreads, n * n * sizeof(int)>>>(currentCuda, nextCuda, n, elementsPerThread);

        // Swap the pointers of current and next arrays
        int* temp = currentCuda;
        currentCuda = nextCuda;
        nextCuda = temp;
    
        // Wait for all threads to finish
        hipDeviceSynchronize();
    }
    hipMemcpy(current, currentCuda, n * n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(next, nextCuda, n * n * sizeof(int), hipMemcpyDeviceToHost);
    /* Print the matrix
    printf("|");
    for (int i = 0; i < n * n; i++) {
        if (current[i] == 1){
            printf(" %d|", current[i]);
        } 
        else{
            printf("%d|", current[i]);
        }
        if (((i+1)%n)==0 && i!=0 && i!=n*n-1) printf("\n|");
        if (((i+1)%(n*n))==0 && i!=0) printf("\n----------------\n");
    }*/
    // Free the memory
    free(current);
    free(next);
    hipFree(currentCuda);
    hipFree(nextCuda);
}

int main() {
    int n; // Size of the Ising model
    int k; // Number of iterations
    int numThreads; // Number of threads
    printf("Enter the size of the Ising model: ");
    scanf("%d", &n);
    printf("Enter the number of iterations: ");
    scanf("%d", &k);
    printf("Enter the number of threads: ");
    scanf("%d", &numThreads);
    
    clock_t start, end;
    double cpu_time_used;

    start = clock();

    isingSimulation(n, k, numThreads);
    
    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time taken: %f seconds\n", cpu_time_used);
    return 0;
}